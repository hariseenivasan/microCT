#include "hip/hip_runtime.h"
/* Program to apply Laplacian of Guassian edge detection on BMP image */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

//Open GL libraries
#include <GL/glew.h>
#ifdef __APPLE__
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#endif
#include "stack.h"
#include "convolution.h"
#include "Timer.h"
#include<list>
const float  PI = 3.14159265358979f;
using namespace std;
struct ImageData{
    GLint   x;      // resolution X
    GLint   y;      // resolution Y
    GLenum  format; // data format (RGB or INDEX..)
    GLenum  type;   // data type (8bit, 16bit or 32bit..)
    GLvoid  *buf;   // image pixel bits
};

// Function declerations
void display (void);

void reshape(int nw, int nh);

void keybord(unsigned char key, int x, int y);

bool initSharedMem();
void clearSharedMem();
bool loadRawImage(char *fileName, int x, int y, unsigned char *data);
void drawString(const char *str, int x, int y, void *font);
void create_LoG();

// constants
const char *FILE_NAME = "lena.bmp";
const int  IMAGE_X = 512;
const int  IMAGE_Y = 512;
const int  MAX_NAME = 1024;
const int  THRESHOLD = 120;

// global variables ////////////////
ImageData       *image;
unsigned char   *inBuf;
unsigned char   *outBuf1;
unsigned char   *outBuf2;
float		*kernel;
unsigned char    *grayscale_Image;
char            fileName[MAX_NAME];
int             imageX;
int             imageY;
int 		kernel_size;
float		sigma_value;
int		threshold;
void            *font = GLUT_BITMAP_8_BY_13;
int             fontWidth = 8;
int             fontHeight = 13;
int             mainWin, subWin1, subWin2, subWin3, subWin4;
double          time1, time2;
int 		kernel_use;

//#define CPU_COMP 1

#define IDX_1D(col, row, stride) ((col * stride) + row)
#define COL_2D(index, stride) (index / stride)
#define ROW_2D(index, stride) (index % stride)
#define ROUNDUP32(integer) ( ((integer-1)/32 + 1) * 32 )

#define SHMEM 8192
#define FLOAT_SZ sizeof(float)


#define CREATE_CONVOLUTION_VARIABLES(psfColRad, psfRowRad) \
	\
const int cornerCol = blockDim.x*blockIdx.x; \
const int cornerRow = blockDim.y*blockIdx.y; \
const int globalCol = cornerCol + threadIdx.x; \
const int globalRow = cornerRow + threadIdx.y; \
const int globalIdx = IDX_1D(globalCol, globalRow, imgRows); \
\
const int localCol = threadIdx.x; \
const int localRow = threadIdx.y; \
const int localIdx = IDX_1D(localCol, localRow, blockDim.y); \
const int localPixels = blockDim.x*blockDim.y; \
\
const int padRectStride = blockDim.y + 2*psfRowRad; \
const int padRectCol = localCol + psfColRad; \
const int padRectRow = localRow + psfRowRad; \
/*const int padRectIdx = IDX_1D(padRectCol, padRectRow, padRectStride); */ \
const int padRectPixels = padRectStride * (blockDim.x + 2*psfColRad); \
\
__shared__ char sharedMem[SHMEM]; \
float* shmPadRect = (float*)sharedMem; \
float* shmOutput = (float*)&shmPadRect[ROUNDUP32(padRectPixels)]; \
int nLoop;

#define PREPARE_PADDED_RECTANGLE(psfColRad, psfRowRad) \
	\
nLoop = (padRectPixels/localPixels)+1; \
for(int loopIdx=0; loopIdx<nLoop; loopIdx++) \
{ \
	int prIndex = loopIdx*localPixels + localIdx; \
	if(prIndex < padRectPixels) \
	{ \
		int prCol = COL_2D(prIndex, padRectStride); \
		int prRow = ROW_2D(prIndex, padRectStride); \
		int glCol = cornerCol + prCol - psfColRad; \
		int glRow = cornerRow + prRow - psfRowRad; \
		int glIdx = IDX_1D(glCol, glRow, imgRows); \
		if(glRow >= 0 && \
				glRow < imgRows && \
				glCol >= 0 && \
				glCol < imgCols) \
		shmPadRect[prIndex] = imgInPtr[glIdx]; \
		else \
		shmPadRect[prIndex] = 0.0f; \
	} \
} \

#define COPY_LIN_ARRAY_TO_SHMEM(srcPtr, dstPtr, nValues) \
	nLoop = (nValues/localPixels)+1; \
for(int loopIdx=0; loopIdx<nLoop; loopIdx++) \
{ \
	int prIndex = loopIdx*localPixels + localIdx; \
	if(prIndex < nValues) \
	{ \
		dstPtr[prIndex] = srcPtr[prIndex]; \
	} \
}

// CUDA kernel to calculate 2d convolution
__global__ void convolve_GPU(unsigned char* imgInPtr,unsigned char* imgOutPtr,float* imgPsfPtr,int imgCols,
		int imgRows,int psfColRad,int psfRowRad)
{

	//Get row and column values
	CREATE_CONVOLUTION_VARIABLES(psfColRad, psfRowRad);

	//define shared memory variable
	shmOutput[localIdx] = 0.0f;

	//calculate image strides
	const int psfStride = psfRowRad*2+1;
	const int psfPixels = psfStride*(psfColRad*2+1);
	float* shmPsf = (float*)&shmOutput[ROUNDUP32(localPixels)];

	COPY_LIN_ARRAY_TO_SHMEM(imgPsfPtr, shmPsf, psfPixels);

	PREPARE_PADDED_RECTANGLE(psfColRad, psfRowRad);

	__syncthreads();

	float accumFloat = 0.0f;

	//compute convolution
	for(int coff=-psfColRad; coff<=psfColRad; coff++)
	{
		for(int roff=-psfRowRad; roff<=psfRowRad; roff++)
		{
			int psfCol = psfColRad - coff;
			int psfRow = psfRowRad - roff;
			int psfIdx = IDX_1D(psfCol, psfRow, psfStride);
			float psfVal = shmPsf[psfIdx];

			int shmPRCol = padRectCol + coff;
			int shmPRRow = padRectRow + roff;
			int shmPRIdx = IDX_1D(shmPRCol, shmPRRow, padRectStride);
			accumFloat += psfVal * shmPadRect[shmPRIdx];
		}
	}

	//apply zero-crossings
	if(accumFloat < 0)
		accumFloat = 0.0f;
	else if(accumFloat > 255.0f)
		accumFloat = 255.0f;
	
	shmOutput[localIdx] = accumFloat;
	__syncthreads();

	imgOutPtr[globalIdx] = shmOutput[localIdx];
}
/// Window Functions
short wire = TRUE;
 
float w, h, tip = 0, turn = 0;
 
float ORG[3] = {0,0,0};
 
float XP[3] = {1,0,0}, XN[3] = {-1,0,0},
      YP[3] = {0,1,0}, YN[3] = {0,-1,0},
      ZP[3] = {0,0,1}, ZN[3] = {0,0,-1};
//Screen constants
const int SCREEN_WIDTH = 900;
const int SCREEN_HEIGHT = 900;
const double CUBESIZE = 1.0;
const double div_num = 1.0;
static GLuint textureID = 0;





 
void reshape (int nw, int nh)
{
    w = nw;
    h = nh;
}
 
void Keybord (unsigned char key, int x, int y)
{
    switch (key) {
       case  'w' : wire = !wire;  break;
       case   27 : exit (0);
       default   : printf ("   %c == %3d from Keybord\n", key, key);
    }
}
 
void Special (int key, int x, int y)
{
    switch (key) {
       case  GLUT_KEY_RIGHT: turn += 5;  break;
       case  GLUT_KEY_LEFT : turn -= 5;  break;
       case  GLUT_KEY_UP   : tip  -= 5;  break;
       case  GLUT_KEY_DOWN : tip  += 5;  break;
 
       default : printf ("   %c == %3d from Special\n", key, key);
    }
}
int clicked=0;
//MouseButton Callback function

void MouseMotion(int x, int y){
	static float lastx=0.0;
	static float lasty=0.0;
	printf("\n%d %d",x,y);
	lastx=(float)x-lastx;
	lasty=(float)y-lasty;
	if( (float)x>lastx)
		turn -=lastx;
	else
		turn+=lastx;
	if((float)y>lasty)
	tip +=lastx;
	else
	tip-=lastx;

	if(abs((int)lastx)>10||(abs((int)lasty)>10))
	{
	lastx=(float)x;
	lasty=(float)y;
	return;
	}
	//glutPostRedisplay();
}
void MouseButton (int btn, int state, int x, int y)
{
//	(state == GLUT_DOWN)?clicked=1:clicked=0;
    //mouse wheel events
    if (btn == 3 || btn == 4 )
    {
        //scroll up
        if(state == GLUT_UP)
           tip+=5;
        
        //scroll down
        else if(state == GLUT_DOWN)
            tip-=5;
    }
    
    //left mouse buttom click
   // else if (btn == GLUT_LEFT_BUTTON && clicked)
   //   (x!=0 && y!=0)?tip+=:  (x!=0 && y==0)?tip += x:((y!=0 && x==0)?tip+=y:tip=tip);
		
    
    //right mosue button click
   // else if (btn == GLUT_RIGHT_BUTTON && clicked)
   //     turn -= x;
    
    //middle mouse button click
   // else if (btn == GLUT_MIDDLE_BUTTON && state ==GLUT_DOWN)
    //    turn -= y;
        
    // Request display update
   // glutPostRedisplay();
    
}

void Draw_Axes (void)
{ 
    glPushMatrix ();
 
       glTranslatef (-2.4, -1.5, -5);
       glRotatef    (0 , 1,0,0);
       glRotatef    (0, 0,1,0);
       glScalef     (0.25, 0.25, 0.25);
 
       glLineWidth (2.0);
 
       glBegin (GL_LINES);
          glColor4f (1,0,0,0);  glVertex3fv (ORG);  glVertex3fv (XP);    // X axis is red.
          glColor4f (0,1,0,0);  glVertex3fv (ORG);  glVertex3fv (YP);    // Y axis is green.
          glColor4f (0,0,1,0);  glVertex3fv (ORG);  glVertex3fv (ZP);    // z axis is blue.
       glEnd();
 
   glPopMatrix ();
}
typedef struct {
unsigned int r:8,g:8,b:8,a:8;
}Color32;
  Color32* lpTex32 ;
  

  typedef struct {GLdouble vertex[3];}vertex3d_Array;
  std::list<vertex3d_Array> v;

void draw_stack ()
{
	int ii; 
	glPushMatrix ();
	 glTranslatef (0, 0, -5);
       glRotatef (tip , 1,0,0);
       glRotatef (turn, 0,1,0);
   float percent=25.0;
	for(ii=0;(CUBESIZE/div_num-((float)ii)*percent/100.0)>= -( CUBESIZE/div_num);ii++){
	//  // cube - TOP
    glBegin(GL_POLYGON);
    glColor4f( 1.0,  1.0,  1.0,  1.0 );
    glTexCoord2f(0.0f, 1.0f);
    glVertex3d(  CUBESIZE/div_num,  CUBESIZE/div_num-((float)ii)*percent/100.0,  CUBESIZE/div_num );
    glTexCoord2f(1.0f, 1.0f);
    glVertex3d(  CUBESIZE/div_num,  CUBESIZE/div_num-((float)ii)*percent/100.0, -CUBESIZE/div_num );
    glTexCoord2f(1.0f, 0.0f);
    glVertex3d( -CUBESIZE/div_num,  CUBESIZE/div_num-((float)ii)*percent/100.0, -CUBESIZE/div_num );
    glTexCoord2f(0.0f, 0.0f);
    glVertex3d( -CUBESIZE/div_num,  CUBESIZE/div_num-((float)ii)*percent/100.0,  CUBESIZE/div_num);
    glEnd();

	}
	GLfloat vertices[][3] = {{1.0,0.5,0.0},{0.4,1.0,0.0},
	{1.0,1.0,0.0}, {1.0,1.0,0.0}, {-1.0,-1.0,1.0}, 
	{1.0,-1.0,1.0}, {1.0,1.0,1.0}, {-1.0,1.0,1.0}};
	int a,b,c,d;
	a=0;b=3;c=2;d=1;
	list<vertex3d_Array> points=v;
	//list<vertex3d_Array>::iterator it;
		//for (it=v.begin(); it != sList.end(); ++it)
//	for(ii = 0; ii<points.size();ii++)
//{
//		
//		vertex3d_Array pt;
//		pt=points.front(); 
//		points.pop_front();
//		//glVertex3f(.0f, 100.0f, -25.0f);
//		//for(int f=0;f<20;f++)
//		{
//		glBegin(GL_LINES);
//		glColor4f( (GLfloat)ii/(float)points.size(),  (float)ii/(float)points.size(),  (float)ii/(float)points.size(),  1.0 );		      
//		glTexCoord2f(0.0f, 1.0f);
//		
//		glVertex3d(pt.vertex[0],pt.vertex[1],-1.0f);
//		glVertex3d(pt.vertex[0],pt.vertex[1],0.0f);
//		glVertex3d(pt.vertex[0],pt.vertex[1],1.0f);
//	
//		glEnd();
//		}
//	}

	
	//glBegin(GL_POINTS);
	//glColor4f( 1.0,  1.0,  1.0,  1.0 );		      
 //   glTexCoord2f(0.0f, 1.0f);
	///*	glColor4fv(colors[a]);*/
	///*	glTexCoord2f(0.0,0.0); */

	//	glVertex3fv(vertices[a]);
	///*	glColor4fv(colors[b]); */
	///*	glTexCoord2f(0.0,1.0); */
	//	glVertex3fv(vertices[b]);
	///*	glColor4fv(colors[c]); */
	///*	glTexCoord2f(1.0,1.0); */
	//	
	//	glVertex3fv(vertices[c]);
	///*	glColor4fv(colors[d]); */
	///*	glTexCoord2f(1.0,0.0); */
	//	
	//	glVertex3fv(vertices[d]);
	//glEnd();
     glPopMatrix (); 

}

void display (void)
{
    //glutSetWindow(subWin3);
 //   glClear(GL_COLOR_BUFFER_BIT); // clear canvas

	glViewport (0, 0, w, h);
    glClear    (GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	
     glEnable(GL_TEXTURE_2D);
  
 
    // Create OpenGL textures
    glGenTextures(1, &textureID);
    
    // Bind the texture
    glBindTexture(GL_TEXTURE_2D, textureID);
    
	 glTexEnvf(GL_TEXTURE_2D,GL_TEXTURE_ENV_MODE,GL_MODULATE);
  glDepthMask(GL_FALSE);
  glEnable(GL_BLEND);
  glBlendFunc(GL_SRC_ALPHA,GL_ONE_MINUS_SRC_ALPHA);
  glColor4f(1.0,1.0,1.0,0.5);//Replace this alpha for transparency
    // Generate texture
	//glTexImage2D(GL_TEXTURE_2D, 0, image->format, imageX, imageY, 0, image->format, image->type, outBuf2);
   
  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, imageX, imageY, 0, GL_RGBA, GL_UNSIGNED_BYTE, lpTex32);
    
    //Set texture parameters
    glTexParameteri (GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexParameteri (GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    
    //Set texture ID
   // glEnable(GL_TEXTURE_2D);
   // glBindTexture( GL_TEXTURE_2D, textureID );
    
    draw_stack();
	Draw_Axes();
    //free texture
    glDeleteTextures( 1, &textureID );
    textureID = 0;
   // Draw_Teapot ();
	
 
    glutSwapBuffers ();
}

//End of window functions
// main program

/*
int main(int argc, char **argv)
{


	kernel_use = 0;
// use default image file if not specified
    if(argc == 7)
    {
        strcpy(fileName, argv[1]);
        imageX = atoi(argv[2]);
        imageY = atoi(argv[3]);
	threshold = atoi(argv[6]);
	kernel_size = atoi(argv[4]);
	sigma_value = atof(argv[5]);
    }
    else{
        printf("Usage: %s <image-file> <width> <height> <threshold>\n", argv[0]);
        strcpy(fileName, FILE_NAME);
        imageX = IMAGE_X;
        imageY = IMAGE_Y;
	threshold = THRESHOLD;
	kernel_use = 1;
        printf("\nUse default image \"%s\", (%d,%d) with threshold value %d\n", fileName, imageX, imageY, threshold);
    }
	   
    // allocate memory for global variables
    if(!initSharedMem()) 
	    return 0;
    
    // open raw image file
    if(!loadRawImage(fileName, imageX, imageY, grayscale_Image))
    {
        clearSharedMem();            
        return 0;
    } 
    
    
    int bmp_image_size = (imageX * imageY * 3) + 48;
    int gray_size = 0;
    
    for (int i=48;i <= bmp_image_size-1;i+=3) {
        unsigned char b = grayscale_Image[i];
        unsigned char g = grayscale_Image[i+1];
        unsigned char r = grayscale_Image[i+2];
        
        //calculate grayscale value
        int grayscale = ((float)r * 0.35) + ((float) g * 0.54) + ((float)b * 0.11);
            
        inBuf[gray_size] = grayscale;
        gray_size = gray_size + 1;
    }
        
    
    // define 5x5 Gaussian kernel
    //float kernel[25] = { 1/256.0f,  4/256.0f,  6/256.0f,  4/256.0f, 1/256.0f,
    //                     4/256.0f, 16/256.0f, 24/256.0f, 16/256.0f, 4/256.0f,
    //                     6/256.0f, 24/256.0f, 36/256.0f, 24/256.0f, 6/256.0f,
    //                     4/256.0f, 16/256.0f, 24/256.0f, 16/256.0f, 4/256.0f,
    //                     1/256.0f,  4/256.0f,  6/256.0f,  4/256.0f, 1/256.0f };
    
    
    
    //float kernel[25] = { 0.0f, 0.0f, -1.0f, 0.0f, 0.0f,
    //                    0.0f, -1.0f, -2.0f, -1.0f, 0.0f,
    //                    -1.0f, -2.0f, 16.0f, -2.0f, -1.0f,
    //                    0.0f, -1.0f, -2.0f, -1.0f, 0.0f,
    //                    0.0f, 0.0f, -1.0f, 0.0f, 0.0f};
       
    
    //float kernel[9] = {1/9.0f, 1/9.0f, 1/9.0f, 1/9.0f, 1/9.0f, 1/9.0f, 1/9.0f, 1/9.0f, 1/9.0f};
   //float kernel_u[9] = {1.0f, 1.0f, 1.0f, 1.0f, -8.0f, 1.0f, 1.0f, 1.0f, 1.0f};  //-- works good filt  
   float kernel_u[9] = {-1.0f, -1.0f, -1.0f, -1.0f, 8.0f, -1.0f, -1.0f, -1.0f, -1.0f};  //-- works good filt
  
    //calculate LoG kernel
   // create_LoG();

    if(kernel_use == 0) {
    printf("LoG kernel values are:");

    for(int i =0; i < (kernel_size * kernel_size) ; i++) {
	    if ( (i % kernel_size) == 0)
		    printf("\n");
	    printf("%0.2f\t", kernel[i]);
    }

    printf("\n\n");
    }
    
    Timer t;

    // perform convolution on CPU
#ifdef CPU_COMP
    t.start();
    convolve2D(inBuf, outBuf1, imageX, imageY, kernel_u, 3, 3, 100);
    t.stop();
    time1 = t.getElapsedTimeInMilliSec();
    printf("Convolution on CPU: %f ms\n", time1);
#endif

    int image_size = imageX * imageY;
    unsigned char *dev_a, *dev_c;
    float *dev_b;

    //calculate the grid and block dimenions on device
    unsigned int blockDimX = 8; // X ~ COL
    unsigned int blockDimY = 32; // Y ~ ROW
    unsigned int gridDimX = imageX/blockDimX; // X ~ COL
    unsigned int gridDimY = imageY/blockDimY; 

    dim3 GRID( gridDimX, gridDimY, 1);
    dim3 BLOCK( blockDimX, blockDimY, 1);
    
    // allocate memory on the GPU
    hipMalloc((void **) &dev_a, image_size * sizeof(char));
    if(kernel_use ==1)
    	hipMalloc((void **) &dev_b, 9 * sizeof(float));
    else
	    hipMalloc((void **) &dev_b, kernel_size * kernel_size * sizeof(float));
    hipMalloc((void **) &dev_c,  image_size * sizeof(char));

    // copy the arrays to the GPU
    hipMemcpy(dev_a, inBuf, image_size * sizeof(char),hipMemcpyHostToDevice);
    if(kernel_use == 1)
	    hipMemcpy(dev_b, kernel_u, 9 * sizeof(float) ,hipMemcpyHostToDevice);
    else
	    hipMemcpy(dev_b, kernel, kernel_size * kernel_size * sizeof(float) ,hipMemcpyHostToDevice);

    // Launch kernels
    t.start(); 
    if(kernel_use == 1)
	    convolve_GPU<<<GRID, BLOCK>>>(dev_a, dev_c, dev_b, imageX, imageY, 3, 3);
    else
	    convolve_GPU<<<GRID, BLOCK>>>(dev_a, dev_c, dev_b, imageX, imageY, kernel_size, kernel_size);
    t.stop();
    time2 = t.getElapsedTimeInMilliSec();
    printf("Convolution on GPU: %f ms\n", time2);

    // copy the result array back from GPU to CPU
    hipMemcpy(outBuf2, dev_c, image_size * sizeof(char) ,hipMemcpyDeviceToHost);
	
	
	lpTex32=(Color32*)calloc(image_size,sizeof(Color32));
	
	vertex3d_Array vertex; int numberofvertex=1;
	double* vert=(double*)malloc(sizeof(double)*3);

	// Apply the threshold value to the output image and replace black with transparent pixels
	//
	//     8 Connected Neighbours for i,j
	//  (i-1,j) (i+1,j) (i+1,j+1) (i,j+1) (i,j-1) (i-1,j-1) (i+1,J-1) (i-1,j+1)
	//  Check threshold value of each connected neighbour
	//
    for(int i =20; i < imageX;i ++)
	for(int j=20; j<imageY;j++) {
	   
			int iplus,jplus,iminus,jminus;

			((i-1)<0 )?iminus=i:iminus=i-1;
			((i+1)>=imageX)?iplus=i:iplus=i+1;
			((j-1)<0 )?jminus=j:jminus=j-1;
			((j+1)>=imageY)?jplus=j:jplus=j+1;
			//if((abs(outBuf2[iminus*imageX+j])<threshold ) && 
			//	(abs(outBuf2[iplus*imageX+j])<threshold ) &&
			//	(abs(outBuf2[iplus*imageX+jplus])<threshold ) &&
			//	(abs(outBuf2[i*imageX+jplus])<threshold ) &&
			//	(abs(outBuf2[i*imageX+jminus])<threshold ) &&
			//	(abs(outBuf2[iminus*imageX+jminus])<threshold ) &&
			//	(abs(outBuf2[iplus*imageX+jminus])<threshold ) &&
			//	(abs(outBuf2[iminus*imageX+jplus])<threshold ) )
			//
			if(outBuf2[i*imageX+j] < threshold)
			 {

		outBuf2[i*imageX+j] = 0; // We are making the pixel black in grayscale.
		//Construct [ pixel
		lpTex32->a=0; // Pixel is transparent
		lpTex32->r=0;
        lpTex32->g=0;
        lpTex32->b=0;

		}else{
			// Construct opaque while pixel
			//realloc(vertex,sizeof(double)*3);
			//vertex.vertex[0]=((double)i/(double)imageX)-0.5f;
			//vertex.vertex[1]=((double)j/(double)imageY)-0.5f;
			//vertex.vertex[2]=0.0;
			//memcpy(vert,vertex,sizeof(double)*3);
			//v.push_back(vertex);

		 lpTex32->a=0xFF; // Opaque
		 // Specify color value[here its white]
		 lpTex32->r=0xFF;
         lpTex32->g=0xFF;
         lpTex32->b=0xFF;
		 numberofvertex++;
		 
		}

       lpTex32++;
    
	}

	//Take backe the memory location to start of image (Need to replace with formula like in line below)
	for(int i =0; i < image_size;i ++, lpTex32--){}
	
	//lpTex32=lpTex32-((image_size+1)*4);

    //clear CUDA memory
   hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    //Draw the image using OpenGL
  

    glutInitWindowSize     (600, 400);
    glutInitWindowPosition (400, 300);
    glutInitDisplayMode    (GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);
	
    glutCreateWindow       ("Corner Axes");
    glutDisplayFunc        (display);
	glutIdleFunc           (display);
    glutReshapeFunc        (reshape);
    glutKeyboardFunc       (Keybord);
    glutSpecialFunc        (Special);

	glutMouseFunc (MouseButton);
	glutMotionFunc(MouseMotion);
    glClearColor   (0.1, 0.2, 0.1, 1.0);
   glEnable       (GL_DEPTH_TEST);
   glMatrixMode   (GL_PROJECTION);
    gluPerspective (40.0, 1.5, 1.0, 10.0);
    glMatrixMode   (GL_MODELVIEW);


   
    glutMainLoop(); // Start GLUT event-processing loop 
	free(lpTex32);
    return 0;
}
*/



///////////////////////////////////////////////////////////////////////////////
// initialize global variables
///////////////////////////////////////////////////////////////////////////////
bool initSharedMem()
{
    image = new ImageData;
    if(!image)
    {
        printf("ERROR: Memory Allocation Failed.\n");
        return false;
    }

    // allocate input/output buffer
    inBuf = new unsigned char[imageX * imageY];
    grayscale_Image = (unsigned char *) malloc((imageX * imageY * 3) + 48);
    outBuf2 = new unsigned char[imageX * imageY];

    if(kernel_use == 0)
	    kernel = new float[kernel_size * kernel_size];

#ifdef CPU_COMP
    outBuf1 = new unsigned char[imageX * imageY];
    if(!outBuf2)
    {
        printf("ERROR: Memory Allocation Failed.\n");
        return false;
    }

#endif

    if(!inBuf || !outBuf2)
    {
        printf("ERROR: Memory Allocation Failed.\n");
        return false;
    }

    // set image data
    image->x = imageX;
    image->y = imageY;
    image->format = GL_LUMINANCE;
    image->type = GL_UNSIGNED_BYTE;
    image->buf = (GLvoid*)inBuf;

    return true;
}



///////////////////////////////////////////////////////////////////////////////
// clean up shared memory
///////////////////////////////////////////////////////////////////////////////
void clearSharedMem()
{
    delete image;
    delete [] inBuf;
#ifdef CPU_COMP
    delete [] outBuf1;
#endif
    delete [] outBuf2;
    delete [] grayscale_Image;
    delete [] kernel;
}



///////////////////////////////////////////////////////////////////////////////
// load 8-bit RAW image
///////////////////////////////////////////////////////////////////////////////
bool loadRawImage(char *fileName, int x, int y, unsigned char *data)
{
    // check params
    if(!fileName || !data)
        return false;

    FILE *fp;
    if((fp = fopen(fileName, "r")) == NULL)
    {
        printf("Cannot open %s.\n", fileName);
        return false;
    }

    // read pixel data
    fread(data, 1, x * y * 3, fp);
    fclose(fp);

    return true;
}






void create_LoG()
{
	int temp, col, dx, dy, dxSq, dySq;
	double sigma_sqr = sigma_value * sigma_value;
	double sigma_f = pow(sigma_value, 5);
	double firstTerm = 0.0f;
	double secondTerm = 0.0f;
	//double norm = 1/(sqrt(2 * PI));
	double norm = 1/(sqrt(2 * PI) * sigma_f);
	int size = kernel_size * kernel_size;

	//printf("sigma sqaure value size value %0.2f\n", sigma_sqr);

	//double const_val = norm / sigma_f;
	//printf("Const value %0.2f\n",const_val );

	for(int i=0;i< size; i++)
	{
		temp = (int)(i/kernel_size);
		col = i - (kernel_size * temp);
		dy = ((int) (kernel_size/2)) - temp;
		dx = col - ((int)(kernel_size/2));
		//printf("dx value value %d\n", dx);
		//printf("dy value value %d\n", dy);
		dxSq = dx*dx;
		dySq = dy*dy;    
		firstTerm  = (dxSq + dySq - (2*sigma_sqr)) * norm;
		secondTerm = exp(-0.5 * (dxSq + dySq) / sigma_sqr);
		//printf("firstTerm value value %0.2f\n", firstTerm);
		//printf("secondTerm value value %0.2f\n", secondTerm);
		kernel[i] = firstTerm * secondTerm;
		//printf("kernel value value %0.2f\n", kernel[i]);
	}

	return;
}


